
#include <hip/hip_runtime.h>
#include<iostream>

#define SIZE 2048
#define THREADS_PER_BLOCK 512

using std::cout;
using std::endl;

// __global__ indicates that this function runs on the GPU but is
// called from the cpu code.
//
// nvcc splits things up into shit that runs on the device and
// things that run on the host. device=gpu, host=cpu.
__global__ void vectorAdd(int* a, int* b, int* c, int n) {
  // a, b, and c point to device memory now. How do we allocate
  // device memory?
  //
  // Host pointers can be passed to device code but cannot
  // be dereferenced by device code. The same is true for host
  // pointers-- they can't be dereferenced by device code.
  //
  // So basically what you do is you let the GPU do shit in its
  // own memory, then copy everything over at the end.
  //
  // Basic CUDA API for dealing with device memory
  // — cudaMalloc(), cudaFree(), cudaMemcpy()
  // — Similar to malloc(), free(), memcpy()
  //
  // Ok, so we're using threads and blocks. A block is a GPU processor
  // basically and each block has some number of threads. In order
  // to split up the work, we use the threaad index (threadIdx), the
  // block index (blockkIdx), and the number of threads per block
  // (blockDim). If you're still confused, read this whole thing:
  //
  // http://www.nvidia.com/content/gtc-2010/pdfs/2131_gtc2010.pdf
  //
  // Basically the reason threads are exposed as opposed to being hardcore
  // abstracted away is that threads can communicate with each other
  // and shit so they offer some advantages over just splitting everything
  // up by block.
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index > n) {
    return;
  }
  c[index] = b[index] + a[index];
}

int main() {
  int *a, *b, *c;

  // Shit that runs on our device needs to deal with stuff
  // allocated in its own memory space. The pointers below
  // are going to be copies of abc that the device has
  // access to. People usually prefix device-side pointers
  // with d_ as a convention.
  int *d_a, *d_b, *d_c;

  a = (int*)malloc(SIZE * sizeof(int));
  b = (int*)malloc(SIZE * sizeof(int));
  c = (int*)malloc(SIZE * sizeof(int));

  // Just like we use malloc to allocate host-side memory,
  // we use cudaMalloc to allocate device-side memory.
  // cudaMalloc needs a pointer to our array and I think
  // it's because it wants to write 0 to it or something
  // when an error occurs.
  hipMalloc((void**) &d_a, SIZE * sizeof(int));
  hipMalloc((void**) &d_b, SIZE * sizeof(int));
  hipMalloc((void**) &d_c, SIZE * sizeof(int));

  for (int i = 0; i < SIZE; i++) {
    a[i] = i;
    b[i] = i;
    c[i] = 0;
  }

  // We need to copy our host-side arrays into our device-side
  // memory.
  hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice);

  // The brackets allows you to call a function meant to run on a
  // device. The left number is blocks, the right number is threads.
  // Don't ask me what the difference is...
  //
  // This sortof explains but it's still somewhat confusing...
  // http://bit.ly/1A9Pww8
  int NUM_BLOCKS = SIZE / THREADS_PER_BLOCK + 1;
  vectorAdd<<<NUM_BLOCKS, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, SIZE);

  // Now we need to copy the device-side results to the host-side
  // arrays.
  hipMemcpy(a, d_a, SIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, d_b, SIZE*sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);

  double c_sum = 0;
  for (int i = 0; i < SIZE; i++) {
    c_sum += c[i];
  }
  cout << "c arr result was: " << c_sum << endl;
  cout << "EXPECTED: " << (double)SIZE*(SIZE-1) << endl;;
  if (c_sum == (double)SIZE*(SIZE-1)) {
    cout << "TEST PASSED" << endl;
  } else {
    cout << "TEST FAILED" << endl;
  }

  free(a);
  free(b);
  free(c);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  return 0;
}
